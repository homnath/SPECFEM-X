#include "GPU.h"

void start_timing_cuda(hipEvent_t* start,hipEvent_t* stop){
     hipEventCreate(start);
       hipEventCreate(stop);
         hipEventRecord( *start, 0 );
         }

         /* ----------------------------------------------------------------------------------------------- */

         void stop_timing_cuda(hipEvent_t* start,hipEvent_t* stop,const char* info_str){
           float time;
  //           // stops events
               hipEventRecord( *stop, 0 );
                 hipEventSynchronize( *stop );
                   hipEventElapsedTime( &time, *start, *stop );
                     hipEventDestroy( *start );
                       hipEventDestroy( *stop );
                         // user output
                           printf("%s: Execution Time = %f ms\n",info_str,time);
                           }

                           /* ----------------------------------------------------------------------------------------------- */

                           void stop_timing_cuda(hipEvent_t* start,hipEvent_t* stop,const char* info_str,float* t){
                             float time;
                               // stops events
                                 hipEventRecord( *stop, 0 );
                                   hipEventSynchronize( *stop );
                                    hipEventElapsedTime( &time, *start, *stop );
                                       hipEventDestroy( *start );
                                         hipEventDestroy( *stop );
                                           // user output
                                             printf("%s: Execution Time = %f ms\n",info_str,time);

                                               // returns time
                                                 *t = time;
                                                 }

void get_free_memory(double* free_db, double* used_db, double* total_db) {

  // gets memory usage in byte
  size_t free_byte ;
  size_t total_byte ;
  hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
  }

  *free_db = (double)free_byte ;
  *total_db = (double)total_byte ;
  *used_db = *total_db - *free_db ;
  return;
}

