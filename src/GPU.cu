#include "hip/hip_runtime.h"

#include "GPU.h"


__global__ void plus_reduce(realw *v1,realw*v2,int N,realw *total){

int tid =threadIdx.x;
int i =blockIdx.x*blockDim.x +threadIdx.x;

//Eachblockloadsitselementsintosharedmemory
__shared__ realw x[128];


x[tid]=(i<N)?v1[i]:0.0;
x[tid]*=(i<N)?v2[i]:0.0;


__syncthreads();

//Buildsummationtreeoverelements.
for (unsigned int s=1; s<128 ; s *= 2) {
   if (tid % (2*s) == 0) x[tid] += x[tid + s];
   __syncthreads();
    }

//Thread0addsthepartialsumtothetotalsum
if(tid ==0)atomicAdd(total,x[tid]);
}



extern "C"
void FC_FUNC_(gpu_dot_product,
              GPU_DOT_PRODUCT)(long* gpu_pointer, realw * v1, realw * v2, int * size,realw * product) {
  realw * d_v1;
  hipMalloc((void**) &d_v1, *size*sizeof(realw)); 
  hipMemcpy(d_v1,v1,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  realw * d_v2;
  hipMalloc((void**) &d_v2, *size*sizeof(realw));
  hipMemcpy(d_v2,v2,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  
  realw * d_product;
  hipMalloc((void**) &d_product,sizeof(realw));
  hipMemset(d_product,0,sizeof(realw));

  int nthreads =128;
  int nblocks = ceil(*size/nthreads ) + 1;
  plus_reduce<<<nblocks,nthreads>>>(d_v1,d_v2,*size,d_product); 
  hipMemcpy(product,d_product,sizeof(realw),hipMemcpyDeviceToHost);

   hipFree(d_v1);
   hipFree(d_v2);
}


extern "C"
void FC_FUNC_(prepare_gpu,
              PREPARE_GPU)(long* gpu_pointer, realw * h_K, int * NGLL3, int * size){

  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  *gpu_pointer = (long)mp;

  hipMalloc((void**) &mp->K,(*NGLL3)*(*NGLL3)*(*size)*sizeof(realw));
  hipMemcpy(mp->K,h_K,sizeof(realw)*(*size)*(*NGLL3)*(*NGLL3),hipMemcpyHostToDevice);


}
