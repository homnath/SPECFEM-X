#include "hip/hip_runtime.h"

#include "GPU.h"


__global__ void plus_reduce(realw *v1,realw*v2,int N,realw *total){

int tid =threadIdx.x;
int i =blockIdx.x*blockDim.x +threadIdx.x;

//Eachblockloadsitselementsintosharedmemory
__shared__ realw x[128];


//x[tid]=v1[i]*v2[i];
x[tid]=(i<N)?v1[i]:0.0;
x[tid]*=(i<N)?v2[i]:0.0;


__syncthreads();

//Buildsummationtreeoverelements.
for (unsigned int s=1; s<128 ; s *= 2) {
   if (tid % (2*s) == 0) x[tid] += x[tid + s];
   __syncthreads();
    }

//Thread0addsthepartialsumtothetotalsum
if(tid ==0)atomicAdd(total,x[tid]);
}



extern "C"
void FC_FUNC_(gpu_dot_product,
              GPU_DOT_PRODUCT)(long* gpu_pointer, realw * v1, realw * v2, int * size,realw * product) {
  realw * d_v1;
  hipMalloc((void**) &d_v1, *size*sizeof(realw)); 
  hipMemcpy(d_v1,v1,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  realw * d_v2;
  hipMalloc((void**) &d_v2, *size*sizeof(realw));
  hipMemcpy(d_v2,v2,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  
  realw * d_product;
  hipMalloc((void**) &d_product,sizeof(realw));
  hipMemset(d_product,0,sizeof(realw));

  hipblasHandle_t cublas_handle = NULL;
  hipblasCreate(&cublas_handle);
  hipblasDdot(cublas_handle, *size, d_v1, 1, d_v2, 1, d_product);
  hipblasDestroy(cublas_handle);
  // int nthreads =128;
  // int nblocks = ceil(*size/nthreads ) + 1;
  // plus_reduce<<<nblocks,nthreads>>>(d_v1,d_v2,*size,d_product); 
  hipMemcpy(product,d_product,sizeof(realw),hipMemcpyDeviceToHost);

   hipFree(d_v1);
   hipFree(d_v2);
}


extern "C"
void FC_FUNC_(prepare_gpu,
              PREPARE_GPU)(long* gpu_pointer, realw * h_K, int * nedof, int * nelmt, int * h_gdof_elmt, int * neq,realw * f, realw* dprecon, realw* u, realw * r, realw * p,realw * KSP_rtol){

  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  *gpu_pointer = (long)mp;

  mp->nelmt = *nelmt ;
  mp->neq = *neq ;
  mp->nedof = *nedof;

  hipMalloc((void**) &mp->K,(*nedof)*(*nedof)*(*nelmt)*sizeof(realw));
  hipMemcpy(mp->K,h_K,sizeof(realw)*(*nelmt)*(*nedof)*(*nedof),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->gdof_elmt,(*nedof)*(*nelmt)*sizeof(int));
  hipMemcpy(mp->gdof_elmt,h_gdof_elmt,sizeof(int)*(*nelmt)*(*nedof),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->kp,(*neq + 1)*sizeof(realw));
  hipMemset(&mp->kp,0,(*neq + 1)*sizeof(realw));

  hipMalloc((void**) &mp->u,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->u,u,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->f,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->f,f,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->dprecon,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->dprecon,dprecon,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->r,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->r,r,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->p,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->p,p,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->kp,(*neq + 1)*sizeof(realw));
  hipMalloc((void**) &mp->pkp,sizeof(realw));

  hipMalloc((void**) &mp->KSP_rtol,sizeof(realw));
  hipMemcpy(mp->KSP_rtol,KSP_rtol,sizeof(realw),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->rz,sizeof(realw));
  hipMalloc((void**) &mp->beta,sizeof(realw));
  hipMalloc((void**) &mp->pkp,sizeof(realw));

}
