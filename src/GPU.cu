#include "hip/hip_runtime.h"
#include "GPU.h"


__inline__ __device__ realw warpReduceMax(realw v) {

 for (int offset = warpSize/2; offset > 0; offset /= 2) 
    v = max(v,__shfl_xor(v, offset));
  return v;
}

__inline__ __device__ realw blockReduceMax(realw val) {

  static __shared__ realw shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceMax(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceMax(val); //Final reduce within first warp

  return val;
}

__global__ void deviceReducemaxKernel(realw *in, realw* out, int N) {
  realw sum = 0.0;
  //reduce multiple elements per thread
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
       i < N; 
       i += blockDim.x * gridDim.x) {
    if (abs(in[i]) > sum) sum = abs(in[i]);
  }
  sum = blockReduceMax(sum);
  if (threadIdx.x==0)  out[blockIdx.x]=sum;

}

void deviceReduceMax(realw *in, realw* out, int N) {
  int threads = 512;
  int blocks = min((N + threads - 1) / threads, 1024);
  realw * max_temp;
  hipMalloc((void**) &max_temp,sizeof(realw)*blocks);

  deviceReducemaxKernel<<<blocks, threads>>>(in, max_temp, N);
  deviceReducemaxKernel<<<1, 1024>>>(max_temp, max_temp, blocks);
  hipMemcpy(out,max_temp,sizeof(realw),hipMemcpyDeviceToHost);
  hipFree(max_temp);
}


__global__ void plus_reduce(realw *v1,realw*v2,int N,realw *total){

int tid =threadIdx.x;
int i =blockIdx.x*blockDim.x +threadIdx.x;

//Eachblockloadsitselementsintosharedmemory
__shared__ realw x[128];


//x[tid]=v1[i]*v2[i];
x[tid]=(i<N)?v1[i]:0.0;
x[tid]*=(i<N)?v2[i]:0.0;


__syncthreads();

//Buildsummationtreeoverelements.
for (unsigned int s=1; s<128 ; s *= 2) {
   if (tid % (2*s) == 0) x[tid] += x[tid + s];
   __syncthreads();
    }

//Thread0addsthepartialsumtothetotalsum
if(tid ==0)atomicAdd(total,x[tid]);
}


// daxpy like routines
__global__ void vecadd(realw *v1, realw *v2, realw c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // vec1 = vec1 + c * vec2
    if (index < N) v1[index] += c * v2[index];
}

__global__ void vecsub(realw *v1, realw *v2, realw c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // vec1 = vec1 - c * vec2
    if (index < N) v1[index] -= c * v2[index];
}

__global__ void vecmult(realw *v1, realw *v2, realw *v3, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // vec1 = vec2  * vec3
    if (index < N) v3[index] = v1[index] + v2[index];
}

__global__ void vecadd2(realw *v1, realw *v2, realw c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // vec1 = c * vec1 + vec2
    if (index < N) v1[index] = c * v1[index] + v2[index] ;
}

////////////////////////////////////////////////////////////////////////////////////////////////////


extern "C"
void FC_FUNC_(gpu_dot_product,
              GPU_DOT_PRODUCT)(long* gpu_pointer, realw * v1, realw * v2, int * size,realw * product) {
  realw * d_v1;
  hipMalloc((void**) &d_v1, *size*sizeof(realw)); 
  hipMemcpy(d_v1,v1,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  realw * d_v2;
  hipMalloc((void**) &d_v2, *size*sizeof(realw));
  hipMemcpy(d_v2,v2,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  
  realw * d_product;
  hipMalloc((void**) &d_product,sizeof(realw));
  hipMemset(d_product,0,sizeof(realw));

  hipblasHandle_t cublas_handle = NULL;
  hipblasCreate(&cublas_handle);
  hipblasDdot(cublas_handle, *size, d_v1, 1, d_v2, 1, d_product);
  hipblasDestroy(cublas_handle);
  // int nthreads =128;
  // int nblocks = ceil(*size/nthreads ) + 1;
  // plus_reduce<<<nblocks,nthreads>>>(d_v1,d_v2,*size,d_product); 
  hipMemcpy(product,d_product,sizeof(realw),hipMemcpyDeviceToHost);

   hipFree(d_v1);
   hipFree(d_v2);
}


extern "C"
void FC_FUNC_(gpu_daxpy_1,
              GPU_DAXPY_1)(long* gpu_pointer, realw * v1, realw * v2, realw * scalar, int * size) {
  
  // Takes in two vectors and a scalar and recomputes the first vector:
  // v1 = v1 + v2*scalar 
  
  realw * d_v1;
  hipMalloc((void**) &d_v1, *size*sizeof(realw)); 
  hipMemcpy(d_v1,v1,sizeof(realw)*(*size),hipMemcpyHostToDevice);
  realw * d_v2;
  hipMalloc((void**) &d_v2, *size*sizeof(realw));
  hipMemcpy(d_v2,v2,sizeof(realw)*(*size),hipMemcpyHostToDevice);

  int nthreads =128;
  int nblocks = ceil(*size/nthreads ) + 1;
  vecadd<<<nblocks,nthreads>>>(d_v1,d_v2,* scalar,*size); 
  hipMemcpy(v1,d_v1,sizeof(realw)*(*size),hipMemcpyDeviceToHost);

  hipFree(d_v1);
  hipFree(d_v2);
}




__global__ void get_p_loc_vector(int *gdof_elmt,realw *p,realw * p_loc){
int tid =threadIdx.x;
int ivec = blockIdx.x;
int nedof = blockDim.x;

p_loc[ivec * nedof + tid] = p[gdof_elmt[ivec * nedof + tid]];
}

__global__ void assemble_kp_vector(int *gdof_elmt,realw *kp,realw * kp_loc){
int tid =threadIdx.x;
int ivec = blockIdx.x;
int nedof = blockDim.x;

//kp[gdof_elmt[tid]] += kp_loc[tid];
atomicAdd(&kp[gdof_elmt[ivec * nedof + tid]],kp_loc[ivec * nedof + tid]);

}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////

extern "C"
void FC_FUNC_(compute_matvec_prod,
              COMPUTE_MATVEC_PROD)(long* gpu_pointer, realw * h_p, realw * h_kp){
//  hipEvent_t start,stop;
//  start_timing_cuda(&start,&stop);
  Mesh* mp = (Mesh*)(*gpu_pointer); //get mesh pointer out of fortran integer container

   // Cuda timing
  hipMemcpy(mp->p,h_p,sizeof(realw)*(mp->neq+1),hipMemcpyHostToDevice);
//  float time;
//  stop_timing_cuda(&start,&stop,"first memcpy",&time);
//  start_timing_cuda(&start,&stop);
  int N = mp->nelmt ;

//  stop_timing_cuda(&start,&stop,"malloc",&time);
//  start_timing_cuda(&start,&stop);
  hipMemset(mp->kp,0,(mp->neq + 1)*sizeof(realw));
  int nthreads = 256;
  int nblocks = ceil((mp->neq + 1)/nthreads) + 1 ;
 // set_vec_zero<<<nblocks, nthreads>>>(mp->kp, (mp->neq + 1));
///  stop_timing_cuda(&start,&stop,"after memset",&time);
  // Cuda timing
//  start_timing_cuda(&start,&stop);
   nthreads = mp->nedof;
  int nblock = N;

  //get_p_loc_vector<<<nblock,nthreads>>>(mp->gdof_elmt + ielm * mp->nedof, mp->p,p_loc);
  const double beta = 0.0;
  const double alpha = 1.0; 
  
  int &m = mp->nedof;
  int n = 1;
  int &k = mp->nedof;
  int &lda = m;
  int &ldb = k;
  int &ldc = m;
  realw * A = mp->K;
  double * &B = mp->p_loc;
  double * &C = mp->kp_loc;
//  stop_timing_cuda(&start,&stop,"in between",&time);
  // Cuda timing
//  start_timing_cuda(&start,&stop);

  
  get_p_loc_vector<<<nblock,nthreads>>>(mp->gdof_elmt, mp->p, B);
//  stop_timing_cuda(&start,&stop,"p_loc",&time);
// start_timing_cuda(&start,&stop);

  hipblasDgemmStridedBatched(mp->cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, mp->nedof * mp->nedof, B, ldb, mp->nedof, &beta, C, ldc, mp->nedof, mp->nelmt);

//  stop_timing_cuda(&start,&stop,"cublas",&time);
// start_timing_cuda(&start,&stop);

  assemble_kp_vector<<<nblock,nthreads>>>(mp->gdof_elmt, mp->kp, C);

//  stop_timing_cuda(&start,&stop,"assemble",&time);
//  start_timing_cuda(&start,&stop);
   //printf("finished loop\n");
realw max_temp;
deviceReduceMax( mp->kp, &max_temp, mp->neq + 1); 
  hipMemcpy(h_kp,mp->kp,sizeof(realw)*(mp->neq+1),hipMemcpyDeviceToHost);


printf("max from kernel : %lf\n",max_temp);

//  stop_timing_cuda(&start,&stop,"memcpy",&time);

}

extern "C"
void FC_FUNC_(prepare_gpu,
              PREPARE_GPU)(long* gpu_pointer, realw * h_K, int * nedof, int * nelmt, int * h_gdof_elmt, int * neq,realw * f, realw* dprecon, realw* u, realw * r, realw * p,realw * KSP_rtol){

  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  *gpu_pointer = (long)mp;

  mp->nelmt = *nelmt ;
  mp->neq = *neq ;
  mp->nedof = *nedof;


  hipblasCreate(&mp->cublas_handle);

  hipMalloc((void**) &mp->K,(*nedof)*(*nedof)*(*nelmt)*sizeof(realw));
  hipMemcpy(mp->K,h_K,sizeof(realw)*(*nelmt)*(*nedof)*(*nedof),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->gdof_elmt,(*nedof)*(*nelmt)*sizeof(int));
  hipMemcpy(mp->gdof_elmt,h_gdof_elmt,sizeof(int)*(*nelmt)*(*nedof),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->kp,(*neq + 1)*sizeof(realw));

  int nthreads = 256;
  int nblocks = ceil((mp->neq + 1)/nthreads) + 1 ;
  hipMemset(mp->kp,0,(mp->neq + 1)*sizeof(realw));


  hipMalloc((void**) &mp->u,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->u,u,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->f,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->f,f,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->dprecon,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->dprecon,dprecon,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->r,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->r,r,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->p,(*neq + 1)*sizeof(realw));
  hipMemcpy(mp->p,p,sizeof(realw)*(*neq+1),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->kp,(*neq + 1)*sizeof(realw));
  hipMalloc((void**) &mp->pkp,sizeof(realw));

  hipMalloc((void**) &mp->p_loc,(*nedof*(*nelmt))*sizeof(realw));
  hipMalloc((void**) &mp->kp_loc,(*nedof*(*nelmt))*sizeof(realw));

  hipMalloc((void**) &mp->KSP_rtol,sizeof(realw));
  hipMemcpy(mp->KSP_rtol,KSP_rtol,sizeof(realw),hipMemcpyHostToDevice);

  hipMalloc((void**) &mp->rz,sizeof(realw));
  hipMalloc((void**) &mp->beta,sizeof(realw));
  hipMalloc((void**) &mp->pkp,sizeof(realw));

}
